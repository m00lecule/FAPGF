#include "hip/hip_runtime.h"
#include <stdio.h>


__global__
void calculate_distances(double r_d, int radix,bool* contaminated,double* R,double* G,double* B,int height,int width){
/*
  __shared__ double buffor_r[5][256+4];
  __shared__ double buffor_g[5][256+4];
  __shared__ double buffor_b[5][256+4];

  int len=256;

  for(int k=0 ; k<=2*radix ; ++k)
    for(int i=threadIdx.x ; i<len+2*radix ; i+=blockDim.x){
      buffor_r[k][i] = 0;
      buffor_g[k][i] = 0;
      buffor_b[k][i] = 0;
    }

  __syncthreads();

  for(int j=0 ; j<=2*radix ; ++j)
    for(int i=threadIdx.x ; i<len+2*radix ; i+=blockDim.x)
      if(i+blockDim.x*blockIdx.x<width){
        buffor_r[j][i] = R[j*width+(blockIdx.x*blockDim.x)+i];
        buffor_g[j][i] = G[j*width+(blockIdx.x*blockDim.x)+i];
        buffor_b[j][i] = B[j*width+(blockIdx.x*blockDim.x)+i];
      }

  __syncthreads();

//wysokosc zawiera ramke
for(int j=radix ; j<height-radix ; ++j){

      for(int y=2*radix ; y>=1 ; --y)
        for(int i=threadIdx.x ; i<len+2*radix ; i+=blockDim.x)
          if(i+blockDim.x*blockIdx.x<width){
            buffor_r[y][i] = buffor_r[y-1][i];
            buffor_g[y][i] = buffor_g[y-1][i];
            buffor_b[y][i] = buffor_b[y-1][i];
          }
      __syncthreads();

      for(int i=threadIdx.x ; i<len+2*radix ; i+=blockDim.x)
        if(i+blockDim.x*blockIdx.x<width){
          buffor_r[0][i] = R[(j+radix)*width+(blockIdx.x*blockDim.x)+i];
          buffor_g[0][i] = G[(j+radix)*width+(blockIdx.x*blockDim.x)+i];
          buffor_b[0][i] = B[(j+radix)*width+(blockIdx.x*blockDim.x)+i];
        }

    __syncthreads();

    if(blockDim.x*blockIdx.x+threadIdx.x+radix<width-2*radix){
      double current_distance;
      int g_n=0;

      for(int y=0 ; y<=2*2 ; y++){
        for(int x=0 ; x<=2*2 ; x++){
          current_distance=0;
          current_distance += (buffor_r[radix][threadIdx.x+radix]-buffor_r[y][threadIdx.x+x])*(buffor_r[radix][threadIdx.x+radix]-buffor_r[y][threadIdx.x+x]);
          current_distance += (buffor_g[radix][threadIdx.x+radix]-buffor_g[y][threadIdx.x+x])*(buffor_g[radix][threadIdx.x+radix]-buffor_g[y][threadIdx.x+x]);
          current_distance += (buffor_b[radix][threadIdx.x+radix]-buffor_b[y][threadIdx.x+x])*(buffor_b[radix][threadIdx.x+radix]-buffor_b[y][threadIdx.x+x]);

          current_distance=sqrt(current_distance);
          //counter nie posiada bufforu zzew
          if(current_distance<r_d)
            g_n++;
        }
      }

      if(g_n<3)
        contaminated[j*width+(blockDim.x*blockIdx.x)+threadIdx.x+radix]=true;
      else
        contaminated[j*width+(blockDim.x*blockIdx.x)+threadIdx.x+radix]=false;
    }
    __syncthreads();
  }

  */
    int counter=0;
    if(blockIdx.x*blockDim.x+threadIdx.x<width-radix&&blockIdx.x*blockDim.x+threadIdx.x>=radix&&blockIdx.y*blockDim.y+threadIdx.y>=radix&&blockIdx.y*blockDim.y+threadIdx.y<height-radix)
    {
        double dist;
        counter=0;
        for(int i=-radix;i<=radix;++i){
            for(int j=-radix;j<=radix;++j){
                dist=sqrt((R[blockIdx.x*blockDim.x+threadIdx.x+width*blockIdx.y*blockDim.y+threadIdx.y]-R[blockIdx.x*blockDim.x+threadIdx.x+i+width*(blockIdx.y*blockDim.y+threadIdx.y+j)])*(R[blockIdx.x*blockDim.x+threadIdx.x+width*blockIdx.y*blockDim.y+threadIdx.y]-R[blockIdx.x*blockDim.x+threadIdx.x+i+width*(blockIdx.y*blockDim.y+threadIdx.y+j)])+(G[blockIdx.x*blockDim.x+threadIdx.x+width*blockIdx.y*blockDim.y+threadIdx.y]-G[blockIdx.x*blockDim.x+threadIdx.x+i+width*(blockIdx.y*blockDim.y+threadIdx.y+j)])*(G[blockIdx.x*blockDim.x+threadIdx.x+width*blockIdx.y*blockDim.y+threadIdx.y]-G[blockIdx.x*blockDim.x+threadIdx.x+i+width*(blockIdx.y*blockDim.y+threadIdx.y+j)])+(B[blockIdx.x*blockDim.x+threadIdx.x+width*blockIdx.y*blockDim.y+threadIdx.y]-B[blockIdx.x*blockDim.x+threadIdx.x+i+width*(blockIdx.y*blockDim.y+threadIdx.y+j)])*(B[blockIdx.x*blockDim.x+threadIdx.x+width*blockIdx.y*blockDim.y+threadIdx.y]-B[blockIdx.x*blockDim.x+threadIdx.x+i+width*(blockIdx.y*blockDim.y+threadIdx.y+j)]));

                if(dist<r_d)
                counter++;
            }
        }

        if(counter<3)
            contaminated[blockIdx.x*blockDim.x+threadIdx.x+width*blockIdx.y*blockDim.y+threadIdx.y]=true;
         else
            contaminated[blockIdx.x*blockDim.x+threadIdx.x+width*blockIdx.y*blockDim.y+threadIdx.y]=false;

    }
}

void gpu_calcul(double r_d,int delay, double* r,double* g, double* b, int w, int h, bool*& noise_map){

  if(r!=nullptr){
    double *p_r, *p_g,*p_b;
    bool* noise_map_cuda;

    //tylko do których zapisujemy osobno 16 streamów moze byc
    //cuda callback
  //  cuda_stream_t stream;

    //cudaStreamCreate(&stream);
    hipMalloc((void**)&p_r,sizeof(double)*w*h);
    hipMalloc((void**)&p_g,sizeof(double)*w*h);
    hipMalloc((void**)&p_b,sizeof(double)*w*h);
    hipMalloc((void**)&noise_map_cuda,sizeof(bool)*w*h);

    //asynchronicznie
    hipMemcpy(p_r,r,sizeof(double)*w*h,hipMemcpyHostToDevice);
    hipMemcpy(p_g,g,sizeof(double)*w*h,hipMemcpyHostToDevice);
    hipMemcpy(p_b,b,sizeof(double)*w*h,hipMemcpyHostToDevice);
    //asynchroicznie

    //calculate_distances<<<((w-2*delay+255)/256),256>>>( r_d , 2 , noise_map_cuda , p_r , p_g, p_b , h , w );

    dim3 Grid((w+7)/8,(h+7)/8);
    dim3 Block(8,8);
    calculate_distances<<<Grid,Block>>>( r_d , 2 , noise_map_cuda , p_r , p_g, p_b , h , w );

    hipDeviceSynchronize();

    hipMemcpy(noise_map,noise_map_cuda,sizeof(bool)*w*h,hipMemcpyDeviceToHost);

    hipFree(noise_map_cuda);
    hipFree(p_r);
    hipFree(p_g);
    hipFree(p_b);

  }
}

__global__ void set(float* tab,float* tab2,float* tab3, int w,int h){
    if(threadIdx.x<w*h)
        tab3[threadIdx.x]=tab2[threadIdx.x]+tab[threadIdx.x];
}

void set_zeros(float* tab,float* tab2, float* tab3, int w, int h)
{
    float* ptr1, *ptr2, *ptr3;
    int dev;
    hipGetDeviceCount(&dev);
    tab3[6]=dev;
    hipMalloc((void**)&ptr1,sizeof(float)*w*h);
    tab3[0]=(int)hipGetLastError();
    hipMalloc((void**)&ptr2,sizeof(float)*w*h);
    tab3[1]=(int)hipGetLastError();
    hipMalloc((void**)&ptr3,sizeof(float)*w*h);
    tab3[2]=(int)hipGetLastError();
    hipMemcpy((void*)ptr1,(void*)tab,sizeof(float)*w*h,hipMemcpyHostToDevice);
    tab3[3]=(int)hipGetLastError();
    hipMemcpy((void*)ptr2,(void*)tab2,sizeof(float)*w*h,hipMemcpyHostToDevice);
    tab3[4]=(int)hipGetLastError();
    hipMemset((void*)ptr3, 0, sizeof(float)*w*h);
    tab3[5]=(int)hipGetLastError();
    set<<<w*h,w*h>>>(ptr1,ptr2,ptr3,w,h);

    hipDeviceSynchronize();
    hipMemcpy((void*)tab3 , (void*)ptr3 , w*h*sizeof(float) , hipMemcpyDeviceToHost);

    //for(int i=0;i<w*h;++i)
    //    tab3[i]=ptr3[i];

    hipFree(ptr1);
    hipFree(ptr2);
    hipFree(ptr3);
}


