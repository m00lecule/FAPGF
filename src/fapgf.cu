#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void calculate_distances(double r_d, int radix, bool* contaminated, double* R, double* G, double* B, int height, int width)
{
  extern __shared__ double s[];
  double *buffor_r = s;                       
  double *buffor_g = &buffor_r[(2*radix+1)*(blockDim.x + 2*radix)];
  double *buffor_b = &buffor_g[(2*radix+1)*(blockDim.x + 2*radix)];

//load data  
for(int i = threadIdx.x ; i < blockDim.x + 2 * radix && (i + blockDim.x * blockIdx.x < width); i += blockDim.x )
  for(int j = 0 ; j < 2*radix ; ++j)
  {
    buffor_r[(j + 1)*(2*radix + 1) + i] = R[(i + blockIdx.x*blockDim.x)*height + j];
    buffor_g[(j + 1)*(2*radix + 1) + i] = G[(i + blockIdx.x*blockDim.x)*height + j];
    buffor_b[(j + 1)*(2*radix + 1) + i] = B[(i + blockIdx.x*blockDim.x)*height + j];
  }

  for(int h = radix ; h < height - radix ; ++h)
  {
    //roll rows by one
    for(int i = threadIdx.x ; i < blockDim.x + 2 * radix && (i + blockDim.x * blockIdx.x < width); i += blockDim.x )
      for(int j = 0 ; j < 2*radix ; ++j)
      {
        buffor_r[j*(2*radix + 1) + i] = buffor_r[(j + 1)*(2*radix + 1) + i];
        buffor_g[j*(2*radix + 1) + i] = buffor_g[(j + 1)*(2*radix + 1) + i];
        buffor_b[j*(2*radix + 1) + i] = buffor_b[(j + 1)*(2*radix + 1) + i];
      } 

    //read next row  
    for(int i = threadIdx.x ; i < blockDim.x + 2 * radix && (i + blockDim.x * blockIdx.x < width); i += blockDim.x )
    {
        buffor_r[(2*radix)*(2*radix + 1) + i] = R[(i + blockIdx.x*blockDim.x)*height + (h + radix)];
        buffor_g[(2*radix)*(2*radix + 1) + i] = G[(i + blockIdx.x*blockDim.x)*height + (h + radix)];
        buffor_b[(2*radix)*(2*radix + 1) + i] = B[(i + blockIdx.x*blockDim.x)*height + (h + radix)];
    }
    __syncthreads();

    if( blockDim.x*blockIdx.x + threadIdx.x + radix < width - 2*radix )
    {
      double current_distance;
      int g_n = 0;

      for(int y = 0 ; y <= 2*radix ; ++y)
      {
        for(int x = 0 ; x <= 2*radix; ++x)
        {
          current_distance=0;
          current_distance += pow((buffor_r[radix*(2*radix + 1) + threadIdx.x + radix] - buffor_r[y*(2*radix + 1) + threadIdx.x + x]), 2);
          current_distance += pow((buffor_g[radix*(2*radix + 1) + threadIdx.x + radix] - buffor_g[y*(2*radix + 1) + threadIdx.x + x]), 2);
          current_distance += pow((buffor_b[radix*(2*radix + 1) + threadIdx.x + radix] - buffor_b[y*(2*radix + 1) + threadIdx.x + x]), 2);

          if(current_distance < r_d)
            g_n++;
        }
      }
      if(g_n < 3)
        contaminated[(radix + threadIdx.x + blockIdx.x*blockDim.x)*height + h] = true;
      else
        contaminated[(radix + threadIdx.x + blockIdx.x*blockDim.x)*height + h] = false;
    }
    __syncthreads();
  }
}

void gpu_calcul(double r_d, int radix, double* r, double* g, double* b, int w, int h, bool*& noise_map){

  if(r!=nullptr)
  {
    double *d_r, *d_g, *d_b;
    bool* d_map;
    size_t img_size = w*h * sizeof(double);
    size_t noise_map_size = w*h * sizeof(bool);

    const int threadsPerBlock = 256;
    int blocksPerGrid = (w - 2*radix + threadsPerBlock - 1) / threadsPerBlock; 
    int sharedMemPerBlock = (2*radix + 1) * 3 * (threadsPerBlock + 2*radix) * sizeof(double);
    hipError_t err;

    hipMalloc((void**)&d_r, img_size);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_r (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipMalloc((void**)&d_g, img_size);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_g (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipMalloc((void**)&d_b, img_size);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipMalloc((void**)&d_map, noise_map_size);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_map (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_r, r, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_g, g, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, img_size, hipMemcpyHostToDevice);

    calculate_distances<<<(blocksPerGrid, threadsPerBlock, sharedMemPerBlock >>>( r_d * r_d, radix, d_map, d_r, d_g, d_b, h, w);

    hipDeviceSynchronize();

    err = hipGetLastError();

    if ( err != hipSuccess )
    {
      fprintf(stderr, "Error during kernel call (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    hipMemcpy(noise_map, d_map, noise_map_size, hipMemcpyDeviceToHost);

    hipFree(d_map);
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
  }
}